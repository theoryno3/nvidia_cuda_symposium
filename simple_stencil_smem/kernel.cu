#include "hip/hip_runtime.h"

#include <stdio.h>

#define N 10000
#define RADIUS 5
#define BLOCK 32

__global__ void stencil_1d(int n, double *in, double *out)
{
	/* allocate shared memory */
	__shared__ double temp[BLOCK + 2*(RADIUS)];

	/* calculate global index in the array */
	int gindex = blockIdx.x * blockDim.x + threadIdx.x;
	int lindex = threadIdx.x + RADIUS;

	/* return if my global index is larger than the array size */
	if( gindex >= n ) return;

	/* read input elements into shared memory */
	temp[lindex] = in[gindex];
	
	/* read the beginning and end of the halo */
	/* some of the threads do this portion */
	if( threadIdx.x < RADIUS )
	{
		temp[lindex - RADIUS] = in[gindex - RADIUS];
		temp[lindex + BLOCK] = in[gindex + BLOCK];
	} /* end if */
	
	/* code to handle the boundary conditions */
	if( gindex < RADIUS || gindex >= (n - RADIUS) ) 
	{
		out[gindex] = (double) gindex * ( (double)RADIUS*2 + 1) ;
		return;
	} /* end if */

	
	double result = 0.0;

	for( int i = -(RADIUS); i <= (RADIUS); i++ ) 
	{
		result += temp[lindex + i];
	}

	out[gindex] = result;
	return;
}

int main()
{
    double *in, *out;
	double *d_in, *d_out;
	int size = N * sizeof( double );

	/* allocate space for device copies of in, out */

	hipMalloc( (void **) &d_in, size );
	hipMalloc( (void **) &d_out, size );

	/* allocate space for host copies of in, out and setup input values */

	in = (double *)malloc( size );
	out = (double *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		in[i] = (double) i;
		out[i] = -99.0;
	}

	/* copy inputs to device */

	hipMemcpy( d_in, in, size, hipMemcpyHostToDevice );
	hipMemset( d_out, 0, size );

	/* calculate block and grid sizes */

	dim3 blocks( BLOCK, 1, 1);
	dim3 grids( (N / blocks.x) + 1, 1, 1);

	/* start the timers */

	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );

	/* launch the kernel on the GPU */

	stencil_1d<<< grids, blocks >>>( N, d_in, d_out );

	/* stop the timers */

	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	float elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop );

	printf("Total time for %d elements was %f ms\n", N, elapsedTime );

	/* copy result back to host */

	hipMemcpy( out, d_out, size, hipMemcpyDeviceToHost );

	for( int i = 0; i < N; i++ )
	{
		if( in[i]*( (double)RADIUS*2+1 ) != out[i] ) printf("error in element %d in = %f out %f\n",i,in[i],out[i] );
	} /* end for */

	/* clean up */

	free(in);
	free(out);
	hipFree( d_in );
	hipFree( d_out );
	
	return 0;
} /* end main */
