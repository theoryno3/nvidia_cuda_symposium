#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <stdio.h>

/* macro for index calculations */

#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )

/* matrix size and thread dimensions */

#define SIZE 1024

/* define blocksize X and blocksize Y and blocksize K */

#define TX 16 // Thread block size, x dimension
#define TY 16 // Thread block size, y dimension
#define BK 16 // square block of K size

__global__ void GPU_shmem2(const int m, double const * const a, double const * const b, double *c )
{

/* setup some constanst for later use */

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;
	const int iby = blockIdx.y * TY;
	const int ibx = blockIdx.x * TX;

/* shared memory arrays for A and B */
	/* insert code for proper size of shared memory segments */
	__shared__ double as[ FIXME ][ FIXME ];
	__shared__ double bs[ FIXME ][ FIXME ];
	
/* space for C to be held in register */

	double c_tmp = 0.0 ;

	/* calculate my thread's initial offset into the A and B matrices */
	/* insert code for proper offset into A and B matrices */
	int aoff = INDX( FIXME, FIXME, m );
	int boff = INDX( FIXME, FIXME, m );

	/* main loop over blocks of K */

	for( int Kblock = 0; Kblock < m; Kblock+=BK )
	{

		/* read block of A into shared memory */
		/* insert code for proper indices */
			as[ FIXME ][ FIXME ] = a[ aoff ];

		/* read block of B into shared memory */
		/* insert code for proper indices */
			bs[ FIXME ][ FIXME ] = b[ boff ];

		/* increment A and B offsets for next round of data reads */

		boff += BK;
		aoff += m * BK;

		/* loop to perform the matmult on the blocks */
		/* insert code for proper indices */

#pragma unroll
		for( int k = 0 ; k < BK ; k++ )
		{
			/* insert code for proper indices into the a and b shared matrices */
			c_tmp += as[ FIXME ][ FIXME ] * bs[ FIXME ][ FIXME ];
		}


	} /* end for Kblock */

	/* set C to its proper index into the C matrix */

	int coff = INDX( ibx + tx, iby + ty, m );

	/* write results to the C matrix */

	c[ coff ] = c_tmp;
 
} /* end GPU_shmem2 */


int main( int argc, char *argv[] )
{

    const int size = SIZE;

    fprintf(stdout, "Matrix size is %d\n",size);

    double *h_a, *h_b, *h_c, *h_c1;
    double *d_a, *d_b, *d_c;
 
    size_t numbytes = (size_t ) size * (size_t ) size * sizeof( double );

    h_a = (double *) malloc( numbytes );
    if( h_a == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_b = (double *) malloc( numbytes );
    if( h_b == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_c = (double *) malloc( numbytes );
    if( h_c == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

	h_c1 = (double *) malloc( numbytes );
    if( h_c1 == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

	/* zero out the host memory for C matrices */

    memset( h_c, 0, numbytes );
    memset( h_c1, 0, numbytes );

    fprintf( stdout, "Total memory required is %lf MB\n", 
       3.0 * (double) numbytes / 1000000.0 );

	/* initialize the A and B matrices */

    for( int i = 0; i < size * size; i++ )
    {
      h_a[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
      h_b[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
    }

	/* allocate a, b, c in gpu memory */

    hipMalloc( (void **)&d_a, numbytes );
    hipMalloc( (void **)&d_b, numbytes );
    hipMalloc( (void **)&d_c, numbytes );
	
	/* copy a and b to device */

	hipMemcpy( d_a, h_a, numbytes, hipMemcpyHostToDevice );
    hipMemcpy( d_b, h_b, numbytes, hipMemcpyHostToDevice );

    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate( &handle );

    double alpha = 1.0;
    double beta  = 0.0;

	/* start timers */

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

	/* call CUBLAS dgemm */

hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                 size, size, size,
                 &alpha, 
                 d_a, size,
                 d_b, size,
                 &beta,
                 d_c, size );

	/* stop timers */

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );

	/* print GPU CUBLAS timing information */

    fprintf(stdout, "Total time GPU CUBLAS is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
    
	/* copy C from device to host for error checking */

    hipMemcpy( h_c, d_c, numbytes, hipMemcpyDeviceToHost );

	/* reset C on device to zero */

	hipMemset( d_c, 0, numbytes );

	/* setup grid and block sizes */

	dim3 threads( TX, TY, 1 );
	dim3 blocks( size / TX, size / TY, 1 );

	/* start timers */

	hipEventRecord( start, 0 );

	/* call GPU_naive */

	GPU_shmem2<<< blocks, threads >>> ( size, d_a, d_b, d_c );

	/* stop timers */

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsedTime, start, stop );

	/* print data for GPU naive */

    fprintf(stdout, "Total time GPU SHMEM is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
                  
	/* copy C back to host */
	
	hipMemcpy( h_c1, d_c, numbytes, hipMemcpyDeviceToHost );

    hipblasDestroy( handle );
    hipEventDestroy( start );
    hipEventDestroy( stop );

	/* check CUBLAS versus GPU NAIVE numerical results */

	double temp = 0.0;

	for( int i = 0; i < size * size; i++ )
	{
		temp += ( h_c[i] - h_c1[i] ) * ( h_c[i] - h_c1[i] );
	} /* end for */

	printf("error is %f\n",temp);
	if( temp > 10 ) printf("Error value is suspiciously high!\n");

	/* cleanup */

    hipFree( d_a );
    hipFree( d_b );
	hipFree( d_c );

    free( h_a );
    free( h_b );
    free( h_c );
    free( h_c1 );

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
