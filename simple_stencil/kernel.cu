#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define N 1024
#define RADIUS 5
#define BLOCK 512

__global__ void stencil_1d(int n, double *in, double *out)
{
	/* calculate global index in the array */
	/* insert code to calculate my global index in the array using block and thread build-in variables */
	int gindex = FIXME
	
	/* return if my global index is larger than the array size */
	if( gindex >= n ) return;

	/* code to handle the boundary conditions */
	if( gindex < RADIUS || gindex >= (n - RADIUS) ) 
	{
		out[gindex] = (double) gindex * ( (double)RADIUS*2 + 1) ;
		return;
	} /* end if */

	/* use result as temporary accumulator variable */
	double result = 0.0;
	
	for( int i = gindex-(RADIUS); i <= gindex+(RADIUS); i++ ) 
	{
		/* add the required elements from the array "in" to the temporary variable "result" */ 
		result = FIXME;
	}

	/* store the result in the "out" array */
	out[gindex] = result;
	return;
}

int main()
{
    double *in, *out;
	double *d_in, *d_out;
	int size = N * sizeof( double );

	/* allocate space for device copies of in, out */

	hipMalloc( (void **) &d_in, size );
	hipMalloc( (void **) &d_out, size );

	/* allocate space for host copies of in, out and setup input values */

	in = (double *)malloc( size );
	out = (double *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		in[i] = (double) i;
		out[i] = 0;
	}

	/* copy inputs to device */

	hipMemcpy( d_in, in, size, hipMemcpyHostToDevice );
	hipMemset( d_out, 0, size );

	/* calculate block and grid sizes */

	dim3 blocks( BLOCK, 1, 1);

	/* insert code for proper grid size in X dimension */
	dim3 grids( FIXME, 1, 1);

	/* start the timers */

	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );

	/* launch the kernel on the GPU */

	stencil_1d<<< grids, blocks >>>( N, d_in, d_out );

	/* stop the timers */

	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	float elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop );

	printf("Total time for %d elements was %f ms\n", N, elapsedTime );

	/* copy result back to host */

	hipMemcpy( out, d_out, size, hipMemcpyDeviceToHost );

	for( int i = 0; i < N; i++ )
	{
		if( in[i]*( (double)RADIUS*2+1 ) != out[i] ) printf("error in element %d in = %f out %f\n",i,in[i],out[i] );
	} /* end for */

	/* clean up */

	free(in);
	free(out);
	hipFree( d_in );
	hipFree( d_out );
	
	return 0;
} /* end main */
